#include "hip/hip_runtime.h"
#include "sigmoid.h"
#include <device_matrix.h>
#include <vector>
#include <fstream>
#include <cassert>
#include <cmath>
#include <ctime>
#include <cstdlib>
// nvcc compiler
#include <device_arithmetic.h>
#include <device_math.h>

using namespace std;

typedef device_matrix<float> mat;

Sigmoid::Sigmoid(){
	_weight.resize(1,2);
	_input.resize(1,1);
	_weight.fillwith(0);
}
Sigmoid::Sigmoid(const mat& w){
	_weight=w;
	_input.resize(_weight.getCols()-1,1);
}
Sigmoid::Sigmoid(size_t out_dim, size_t inp_dim){
	_weight.resize(out_dim,inp_dim+1);  // +1 for bias
	rand_init();
	//_weight/=sqrt(inp_dim);
}

Sigmoid::~Sigmoid(){
}

void Sigmoid::forward(mat& out, const mat& in, bool train){
	mat _inp = mat(in);
	pushOne(_inp);
	//fill with 1 for computation simplicity
	out = ext::sigmoid( (_weight * _inp));
	if(train){
		_input = in;
	}
}

// assume error pass through var "delta"
void Sigmoid::backPropagate(mat& out, const mat& delta, float rate){
	assert( (delta.getRows()==_weight.getRows()) && (delta.getCols()==_input.getCols()) );
	mat withoutBias(_weight.getRows(),_weight.getCols()-1);
	CCE(hipMemcpy(withoutBias.getData(),_weight.getData(),withoutBias.size() * sizeof(float),hipMemcpyDeviceToDevice));
	mat _tmp( (~withoutBias) * delta);
	mat one(_input.getRows(),_input.getCols(),1);
	mat diff= (_input) & (one-_input);
	out = diff & _tmp;   // this part need tesing
	// update weight
	mat _inp(_input);
	pushOne(_inp);
	gemm(delta,_inp,_weight,(float)-1.0*rate,(float)1.0,false,true);
	//gemm(delta,_inp,_weight,(float)-1.0*rate/(float)_input.getCols(),(float)1.0,false,true);
}

void Sigmoid::getSigDiff(mat& delta,const mat& error){
	assert( (error.getRows()==_weight.getRows()) && (error.getCols()==_input.getCols()) );
	mat one(_weight.getRows(),_input.getCols(),1);
	mat _inp(_input);
	pushOne(_inp);
	delta = (_weight * _inp);
	mat sig=ext::sigmoid(delta);
	delta = (sig) & (one-sig) & error;
}

void Sigmoid::write(ofstream& out){
	float* h_data = new float[_weight.size()];
	CCE(hipMemcpy( h_data, _weight.getData(), _weight.size() * sizeof(float), hipMemcpyDeviceToHost));
    out<<"\n<sigmoid> "<<_weight.getRows()<<" "<<_weight.getCols()<<endl;
    for(size_t i=0;i<_weight.getRows();++i){
    for(size_t j=0;j<_weight.getCols()-1;++j){
                out<<" "<<h_data[_weight.getRows()*j+i]; 
            }
            out<<endl;
    }
    out<<"<bias> "<<_weight.getRows()<<endl;
    for(size_t t=0;t<_weight.getRows();++t)
                out<<" "<<h_data[_weight.getRows()*(_weight.getCols()-1)+t];
    out<<endl;
	delete [] h_data;
}

void Sigmoid::print(FILE* fid, int precision, char delimiter){
	float* h_data = new float[_weight.size()];
	CCE(hipMemcpy( h_data, _weight.getData(), _weight.size() * sizeof(float), hipMemcpyDeviceToHost));

	char format[16];
	sprintf(format,"%c%%.%de",delimiter,(precision>0)? precision :0);
	
	fprintf(fid,"<sigmoid> %d %d \n",_weight.getRows() ,_weight.getCols()); // <sigmoid> outputDimension inputDimension
	for(size_t i=0;i<_weight.getRows();++i){
		for(size_t j=0;j<_weight.getCols()-1;++j)
			fprintf(fid,format,h_data[j*_weight.getRows()+i]);
		fprintf(fid,"\n");
	}
	
	fprintf(fid,"<bias> %d \n",_weight.getRows()); // <bias> output dimensions
	for(size_t t=0;t<_weight.getRows();++t)
		fprintf(fid,format,h_data[_weight.getRows() * (_weight.getCols()-1) + t]);
	fprintf(fid,"\n");
	
	delete [] h_data;
}
size_t Sigmoid::getInputDim(){
	return _weight.getCols()-1;
}
size_t Sigmoid::getOutputDim(){
	return _weight.getRows();
}
void Sigmoid::rand_init(){
    srand(time(0));
	size_t _s=_weight.size();
	float* h_data = new float [_s];
	for (size_t i=0; i<_s; ++i)
		h_data[i]=(rand() / (float) RAND_MAX) -0.5;
	CCE(hipMemcpy(_weight.getData(), h_data, _weight.size() * sizeof(float), hipMemcpyHostToDevice));
	delete [] h_data;
}

void Sigmoid::pushOne(mat& input){
	device_matrix<float> tmp(~input);
    float* h_data = new float [input.size()+input.getCols()];
	CCE(hipMemcpy(h_data, tmp.getData(), tmp.size() * sizeof(float), hipMemcpyDeviceToHost));
    for(size_t t=0;t<tmp.getRows();++t)
	h_data[tmp.size()+t]=1;
	tmp.resize(tmp.getRows(),tmp.getCols()+1);
	CCE(hipMemcpy(tmp.getData(), h_data, tmp.size() * sizeof(float), hipMemcpyHostToDevice));
    input=~tmp;
	delete [] h_data;
}

