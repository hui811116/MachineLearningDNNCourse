#include "hip/hip_runtime.h"
#include "transforms.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include <cstdlib>
#include <string>
#include <device_matrix.h>
#include <device_arithmetic.h>
#include <device_math.h>
#include "util.h"
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;
using namespace ext;

typedef device_matrix<float> mat;
/////////////helper functions//////////////////////

template<typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
	T C;

	__host__ __device__
	linear_index_to_row_index(T C) : C(C) {}
	
	__host__ __device__
	T operator()(T i)
	{
			return i/C;
	}
};

void substractMaxPerRow(mat& x);
mat getRowMax(mat& C);
__global__ void substract_max_per_row(float* const A,float* const rmax, unsigned int rows , unsigned int cols);

void substractMaxPerRow(mat& x) {
	mat rmax = getRowMax(x);

	const int N = 32;
	dim3 grid;
	grid.x = (unsigned int) ceil((float) x.getCols() / N );
	grid.y = (unsigned int) ceil((float) x.getRows() / N );
	dim3 threads(N,N);

	substract_max_per_row<<<grid, threads>>>(x.getData(),rmax.getData() , x.getRows(),x.getCols());
	CCE(hipDeviceSynchronize());
}


__global__ void substract_max_per_row(float* const A, float * const rmax, unsigned int rows,unsigned int cols){
	int x = blockIdx.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= cols|| y>= rows)
			return;
	A[x * rows +y] -= rmax[y];
}

mat getRowMax(mat& C)
{
	mat rmax(C.getRows(),1);
	mat At = ~C;
	thrust::device_vector<float>row_indices(C.getRows());
	thrust::device_vector<float>row_results(C.getRows());
	thrust::reduce_by_key
	(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C.getCols())),
	 thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C.getCols())) +C.size(),thrust::device_ptr<float>(At.getData()),row_indices.begin(),
	 thrust::device_ptr<float>(rmax.getData()),thrust::equal_to<float>(),thrust::maximum<float>());
	
	return rmax;
}
//////////////////////////////////////////////
///////TRANSFORMS/////////////

Transforms::Transforms(const Transforms& t):_w(t._w),_i(t._i),_pw(t._pw){}

Transforms::Transforms(const mat& w,const mat& b){
	assert(b.getRows()==1 || b.getCols()==1);
	size_t r=b.getRows(),c=b.getCols();
	if(r==1){r=c;c=1;}
	assert(w.getRows()==r);
	float* h_data=new float[w.size()+b.size()];
	float* b_data=new float[b.size()];
	CCE(hipMemcpy(h_data,w.getData(),w.size() *sizeof(float) ,hipMemcpyDeviceToHost));
	CCE(hipMemcpy(b_data,w.getData(),b.size() *sizeof(float) ,hipMemcpyDeviceToHost));
	for(size_t t=0;t<b.size();++t)
			h_data[w.size()+t]=b_data[t];
	_w.resize(w.getRows(),w.getCols()+1);
	CCE(hipMemcpy(_w.getData(),h_data,(w.size()+b.size()) * sizeof(float), hipMemcpyHostToDevice));
	delete [] b_data;
	delete [] h_data;
	_pw.resize(_w.getRows(),_w.getCols(),0);
}

Transforms::Transforms(size_t inputdim,size_t outputdim,float range){
	_w.resize(outputdim,inputdim+1);
	rand_init(_w,range); // uniform distribution
	_w/=sqrt((float)inputdim);
	_pw.resize(outputdim,inputdim+1,0);
}

Transforms::Transforms(size_t inputdim,size_t outputdim,myNnGen& ran){
	_w.resize(outputdim,inputdim+1);
	rand_norm(_w,ran);  // default variance = 0.2 , to change varance head to include/util.h
	_w/=sqrt((float)inputdim);
	_pw.resize(outputdim,inputdim+1,0);
}
size_t Transforms::getInputDim()const{
	return _w.getCols();
}
size_t Transforms::getOutputDim()const{
	return _w.getRows();
}

void Transforms::print(ofstream& out){
	float* h_data = new float[_w.size()];
	CCE(hipMemcpy( h_data, _w.getData(), _w.size() * sizeof(float), hipMemcpyDeviceToHost));
    for(size_t i=0;i<_w.getRows();++i){
    for(size_t j=0;j<_w.getCols()-1;++j){
                out<<" "<<h_data[_w.getRows()*j+i]; 
            }
            out<<endl;
    }
    out<<"<bias> "<<_w.getRows()<<endl;
    for(size_t t=0;t<_w.getRows();++t)
                out<<" "<<h_data[_w.getRows()*(_w.getCols()-1)+t];
	out << endl;
	delete [] h_data;
}
///////////////////////////////
/////////SIGMOID///////////////

Sigmoid::Sigmoid(const Sigmoid& s): Transforms(s){
}
Sigmoid::Sigmoid(const mat& w, const mat& bias): Transforms(w,bias){
}
Sigmoid::Sigmoid(size_t inputdim,size_t outputdim,float range): Transforms(inputdim,outputdim,range){
}
Sigmoid::Sigmoid(size_t inputdim,size_t outputdim,myNnGen& ran): Transforms(inputdim,outputdim,ran){
}
void Sigmoid::forward(mat& out,const mat& in,bool train){
	mat _inp(in);
	pushOne(_inp);
	out=sigmoid(_w * _inp);
	if(train){
		_i=in;
	}
}
void Sigmoid::backPropagate(mat& out,const mat& delta, float rate,float momentum){
	assert( (delta.getRows()==_w.getRows()) && (delta.getCols()==_i.getCols()) );
	mat withoutBias(_w.getRows(),_w.getCols()-1);
	CCE(hipMemcpy(withoutBias.getData(),_w.getData(),withoutBias.size() * sizeof(float),hipMemcpyDeviceToDevice));
	mat one(_i.getRows(),_i.getCols(),1);
	out = _i & (one-_i) & (~withoutBias * delta);   // this part need tesing
	// update weight
	mat _inp(_i);
	pushOne(_inp);
	_pw= delta * ~_inp + _pw * momentum;
	rate/=(float)_i.getCols();
	_w -= _pw * rate;
	//gemm(delta,_inp,_w,(float)-1.0*rate,(float)1.0,false,true);
}
void Sigmoid::write(ofstream& out){
	out<<"<sigmoid> "<<_w.getRows()<<" "<<_w.getCols()-1<<endl;
	print(out);
}

///////////////////////////////
///////////SOFTMAX/////////////

Softmax::Softmax(const Softmax& s): Transforms(s){
}
Softmax::Softmax(const mat& w, const mat& bias):Transforms(w,bias){
}
Softmax::Softmax(size_t inputdim,size_t outputdim,float range): Transforms(inputdim,outputdim,range){
}
Softmax::Softmax(size_t inputdim,size_t outputdim,myNnGen& ran): Transforms(inputdim,outputdim,ran){
}
void Softmax::forward(mat& out,const mat& in,bool train){
	mat inp=in;
	pushOne(inp);
	mat z=~(_w * inp);
	substractMaxPerRow(z);
	z=~z; // transpose to column vectors
	mat p(z.getRows(), z.getCols());
	
	thrust::device_ptr<float> zPtr(z.getData());
	thrust::device_ptr<float> pPtr(p.getData());
	thrust::transform(zPtr, zPtr + z.size(),pPtr, func::exp<float>());

	mat sumOfProb =  (mat(p.getRows(), p.getRows(),0) += 1) * p;
	out.resize(_w.getRows(),in.getCols());
	thrust::device_ptr<float> outptr(out.getData());
	thrust::device_ptr<float> sPtr(sumOfProb.getData());
	thrust::transform(pPtr,pPtr+p.size(), sPtr,outptr,thrust::divides<float>());

	if(train){
		_i=in;
	}
}

void Softmax::backPropagate(mat& out,const mat& delta,float rate, float momentum){
	assert( (delta.getRows()==_w.getRows()) && (delta.getCols()==_i.getCols()) );
	mat withoutBias(_w.getRows(),_w.getCols()-1);
	CCE(hipMemcpy(withoutBias.getData(),_w.getData(),withoutBias.size() * sizeof(float),hipMemcpyDeviceToDevice));
	mat one(_i.getRows(),_i.getCols(),1);
	out = _i & (one-_i) & (~withoutBias * delta);   // this part need tesing
	//update weight
	mat inp(_i);
	pushOne(inp);	
	_pw=delta * ~inp + _pw * momentum;
	rate/=(float)_i.getCols();
	_w-= _pw * rate;
	//gemm(delta,inp,_w,(float)-1.0*rate,(float)1.0,false,true);
	
}
void Softmax::write(ofstream& out){
	out<<"<softmax> "<<_w.getRows()<<" "<<_w.getCols()-1<<endl;
	print(out);
}
///////////////////////////////
