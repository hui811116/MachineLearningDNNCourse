#include <iostream>
#include <vector>
#include <device_matrix.h>

#include <device_arithmetic.h>
#include <device_math.h>

using namespace std;

typedef device_matrix<float> mat;

template <typename T>
void randomInit(device_matrix<T>& m) {
  T* h_data = new T [m.size()];
  for (int i=0; i<m.size(); ++i)
    h_data[i] = rand() / (T) RAND_MAX;
  hipMemcpy(m.getData(), h_data, m.size() * sizeof(T), hipMemcpyHostToDevice);
  delete [] h_data;
}

int main(){

mat A(5,8),B(8,5);
randomInit(A);
randomInit(B);

printf("A=\n");
A.print();
printf("B=\n");
B.print();

printf("A * B= \n"); (A*B).print();

//testing element-wise operation

mat C(8,1), D(8,1);
randomInit(C);
randomInit(D);

printf("C=\n");
C.print();
printf("D=\n");
D.print();

printf("C & D= \n"); (C&D).print();

return 0;
}
