#include <iostream>
#include <vector>
#include <ctime>
#include <cstdlib>
#include <device_matrix.h>

#include <device_arithmetic.h>
#include <device_math.h>

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <cuda_memory_manager.h>
#include "parser.h"
#include "transforms.h"
#include "util.h"


using namespace std;

typedef device_matrix<float> mat;

template <typename T>
void randomInit(device_matrix<T>& m) {
  T* h_data = new T [m.size()];
  for (int i=0; i<m.size(); ++i)
    h_data[i] = rand() / (T) RAND_MAX;
  hipMemcpy(m.getData(), h_data, m.size() * sizeof(T), hipMemcpyHostToDevice);
  delete [] h_data;
}

template <typename T>
void pushOne(device_matrix<T>& m) {
  device_matrix<T> tmp(~m);
  T* h_data = new T [m.size()+m.getCols()];
  hipMemcpy(h_data, tmp.getData(), tmp.size() * sizeof(T), hipMemcpyDeviceToHost);
  tmp.resize(tmp.getRows(),tmp.getCols()+1);
  for(size_t t=0;t<tmp.getRows();++t)
  h_data[m.size()+t]=1;
  hipMemcpy(tmp.getData(), h_data, tmp.size() * sizeof(T), hipMemcpyHostToDevice);
  m=~tmp;
  delete [] h_data;
}

int main(int argc,char** argv){

PARSER p;
p.addOption("--dim",false);
p.read(argc,argv);
string str;
p.getString("--dim",str);
vector<size_t> v;
parseDim(str,v);
for(size_t t=0;t<v.size();++t)
	cout<<" "<<v[t];
cout<<endl;


size_t dim=500;
float n=0.02;
srand(time(0));

mat A(5,8),B(8,5);
randomInit(A);
randomInit(B);

//testing element-wise operation

mat C(8,2), D(8,2,2.5);
randomInit(C);
randomInit(D);

//testing sigmoid function
/*
printf("divide by a const num\n");
C.print();
printf("\n");
(C*n/(float)dim).print();
*/
/*
mat in(10,3);
randomInit(in);
Softmax s1(10,10);
mat out;
s1.forward(out,in,true);
cout<<"out"<<endl;
out.print();
mat bk;
s1.backPropagate(bk,out,0.02,0);
cout<<"bk="<<endl;
bk.print();
*/
/*
Sigmoid s2(10,10);
cout<<"testing sigmoid"<<endl;
s2.forward(out,in,true);
cout<<"out"<<endl;
out.print();
s2.backPropagate(bk,out,0.02,0);
cout<<"bk="<<endl;
bk.print();
*/
cout<<"testing util functions"<<endl;
mat W(5,8);
randomInit(W);
mat B1(5,1,1);
cout<<"W="<<endl;
W.print();
cout<<"calling getBias()"<<endl;
mat tmp;
getBias(tmp,W);
cout<<"result:"<<endl;
tmp.print();
cout<<"calling replaceBias()"<<endl;
replaceBias(W,B1);
cout<<"result"<<endl;
W.print();

return 0;
}
