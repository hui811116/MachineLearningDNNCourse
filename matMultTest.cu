#include <iostream>
#include <vector>
#include <ctime>
#include <cstdlib>
#include <device_matrix.h>

#include <device_arithmetic.h>
#include <device_math.h>
#include "sigmoid.h"

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <cuda_memory_manager.h>
#include "parser.h"


using namespace std;

typedef device_matrix<float> mat;

template <typename T>
void randomInit(device_matrix<T>& m) {
  T* h_data = new T [m.size()];
  for (int i=0; i<m.size(); ++i)
    h_data[i] = rand() / (T) RAND_MAX;
  hipMemcpy(m.getData(), h_data, m.size() * sizeof(T), hipMemcpyHostToDevice);
  delete [] h_data;
}

template <typename T>
void pushOne(device_matrix<T>& m) {
  device_matrix<T> tmp(~m);
  T* h_data = new T [m.size()+m.getCols()];
  hipMemcpy(h_data, tmp.getData(), tmp.size() * sizeof(T), hipMemcpyDeviceToHost);
  tmp.resize(tmp.getRows(),tmp.getCols()+1);
  for(size_t t=0;t<tmp.getRows();++t)
  h_data[m.size()+t]=1;
  hipMemcpy(tmp.getData(), h_data, tmp.size() * sizeof(T), hipMemcpyHostToDevice);
  m=~tmp;
  delete [] h_data;
}

int main(int argc,char** argv){

PARSER p;


srand(time(0));

mat A(5,8),B(8,5);
randomInit(A);
randomInit(B);

//testing element-wise operation

mat C(8,2), D(8,2,2.5);
randomInit(C);
randomInit(D);

//testing sigmoid function

Sigmoid n1(5,5);

C.resize(8,3);
randomInit(C);

printf("testing push one \n");
pushOne(C);
C.print();

printf("testing ext::sigmoid\n");
(ext::sigmoid(C)).print();

n1.print();

printf("minus const num\n");
C.print();
printf("\n");
(C-1).print();

A.resize(5,8);B.resize(5,8);
randomInit(A);randomInit(B);

C.resize(5,5);
randomInit(C);
gemm(A,B,C,(float)-1,(float)1,false,true);
printf("C=\n");
C.print();

return 0;
}
